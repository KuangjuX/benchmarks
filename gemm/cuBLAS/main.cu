#include "hip/hip_runtime.h"
#include "cublas_gemm.cuh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

float rand_float(float a = 1e-3, float b = 1) {
    float random = ((float)rand()) / (float)RAND_MAX;
    float diff = b - a;
    float r = random * diff;
    return a + r;
}

template <const int kM, const int kN, const int kK>
void run_test() {
    using DType = __half;

    thrust::host_vector<DType> h_a(kM * kK);
    for (int i = 0; i < h_a.size(); ++i)
        h_a[i] = static_cast<DType>(rand_float());

    thrust::host_vector<DType> h_b(kK * kN);
    for (int i = 0; i < h_b.size(); ++i)
        h_b[i] = static_cast<DType>(rand_float());

    thrust::host_vector<DType> h_c(kM * kN);
    thrust::fill(h_c.begin(), h_c.end(), 0.);

    thrust::device_vector<DType> d_a = h_a;
    thrust::device_vector<DType> d_b = h_b;
    thrust::device_vector<DType> d_c = h_c;

    const DType* A = thrust::raw_pointer_cast(d_a.data());
    const DType* B = thrust::raw_pointer_cast(d_b.data());
    DType* C = thrust::raw_pointer_cast(d_c.data());

    std::cout << std::fixed << std::setprecision(4)
              << "elapsed time: " << cublas_hgemm<kM, kN, kK>(A, B, C) << " ms"
              << std::endl;
}

int main(int argc, char* argv[]) {
    run_test<128, 128, 128>();

    return 0;
}
